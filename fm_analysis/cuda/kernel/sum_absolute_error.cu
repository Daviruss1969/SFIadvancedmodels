
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void sum_absolute_error(float* golden_fm, float* input_fm, float* result, int N) {
        int lindex = threadIdx.x;
        int gindex = blockDim.x * blockIdx.x + lindex;

        // Compute absolute_difference
        float absolute_difference;
        if (gindex < N) {
            absolute_difference = abs(golden_fm[gindex] - input_fm[gindex]);
        } else {
            absolute_difference = .0f;
        }

        // Declare shared memory
        __shared__ float sharedData[1024];

        // Put data into shared memory
        sharedData[lindex] = absolute_difference;
        __syncthreads();

        // Parallel reduction
        for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
            if (lindex < offset) {
                sharedData[lindex] += sharedData[lindex + offset];
            }
            __syncthreads();
        }

        // Add the results in each block
        if (lindex == 0) {
            atomicAdd(result, sharedData[0]);
        }
    }
}