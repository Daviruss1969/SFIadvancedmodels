
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void sum(float* fm, float* result, int N) {
        int batch_id = blockIdx.y;

        int lindex = threadIdx.x;
        int offset_batch = blockDim.x * blockIdx.x + lindex;

        int gindex = batch_id * N + offset_batch;

        float value = .0f;
        if (offset_batch < N) {
            value = fm[gindex];
        }

        // Declare shared memory
        __shared__ float sharedData[1024];

        // Put data into shared memory
        sharedData[lindex] = value;
        __syncthreads();

        // Parallel reduction
        for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
            if (lindex < offset) {
                sharedData[lindex] += sharedData[lindex + offset];
            }
            __syncthreads();
        }

        // Add the results in each block
        if (lindex == 0) {
            atomicAdd(&result[batch_id], sharedData[0]);
        }
    }
}