
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void sum(float* fm, float* result, int N) {
        int lindex = threadIdx.x;
        int gindex = blockDim.x * blockIdx.x + lindex;

        float value = .0f;
        if (gindex < N) {
            value = fm[gindex];
        }

        // Declare shared memory
        __shared__ float sharedData[1024];

        // Put data into shared memory
        sharedData[lindex] = value;
        __syncthreads();

        // Parallel reduction
        for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
            if (lindex < offset) {
                sharedData[lindex] += sharedData[lindex + offset];
            }
            __syncthreads();
        }

        // Add the results in each block
        if (lindex == 0) {
            atomicAdd(result, sharedData[0]);
        }
    }
}