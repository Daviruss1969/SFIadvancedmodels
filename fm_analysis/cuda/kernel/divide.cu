
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void divide(float* input_1, float* input_2, float* result, int N) {
        int lindex = threadIdx.x;
        int gindex = blockDim.x * blockIdx.x + lindex;

        // Perform the division for each inputs
        if (gindex < N) {
            result[gindex] = input_1[gindex] / input_2[gindex];
        }
    }
}