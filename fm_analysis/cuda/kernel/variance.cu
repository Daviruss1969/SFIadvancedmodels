
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void variance(float* fm, float* result, float mean_value, int N) {
        int lindex = threadIdx.x;
        int gindex = blockDim.x * blockIdx.x + lindex;

        // Compute the square of the difference between one value and the mean value
        float value = .0f;
        if (gindex < N) {
            float diff = fm[gindex] - mean_value;
            value = diff*diff;
        }

        // Declare shared memory
        __shared__ float sharedData[1024];

        // Put data into shared memory
        sharedData[lindex] = value;
        __syncthreads();

        // Parallel reduction
        for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
            if (lindex < offset) {
                sharedData[lindex] += sharedData[lindex + offset];
            }
            __syncthreads();
        }

        // Add the results in each block
        if (lindex == 0) {
            atomicAdd(result, sharedData[0]);
        }
    }
}