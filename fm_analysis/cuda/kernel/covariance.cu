
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void covariance(float* golden_fm, float* input_fm, float* result, float golden_mean_value, float input_mean_value, int N) {
        int batch_id = blockIdx.y;

        int lindex = threadIdx.x;
        int offset_batch = blockDim.x * blockIdx.x + lindex;

        int gindex = batch_id * N + offset_batch;

        // Compute the product of the difference between one value and the mean value corresponding to a feature map
        float diff = .0f;
        if (offset_batch < N) {
            diff = (golden_fm[gindex] - golden_mean_value) * (input_fm[gindex] - input_mean_value);
        }

        // Declare shared memory
        __shared__ float sharedData[1024];

        // Put data into shared memory
        sharedData[lindex] = diff;
        __syncthreads();

        // Parallel reduction
        for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
            if (lindex < offset) {
                sharedData[lindex] += sharedData[lindex + offset];
            }
            __syncthreads();
        }

        // Add the results in each block
        if (lindex == 0) {
            atomicAdd(&result[batch_id], sharedData[0]);
        }
    }
}