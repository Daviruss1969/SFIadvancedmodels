
#include <hip/hip_runtime.h>
extern "C" {
    __global__ void sum_absolute_error(float* golden_fm, float* input_fm, float* result, int N, float order_norm = 1.f) {
        int batch_id = blockIdx.y;

        int lindex = threadIdx.x;
        int offset_batch = blockDim.x * blockIdx.x + lindex;

        int gindex = batch_id * N + offset_batch;

        // Compute absolute_difference
        float absolute_difference = .0f;
        if (offset_batch < N) {
            absolute_difference = abs(golden_fm[gindex] - input_fm[gindex]);

            if (order_norm != 1.f) {
                absolute_difference = pow(absolute_difference, order_norm);
            }
        }

        // Declare shared memory
        __shared__ float sharedData[1024];

        // Put data into shared memory
        sharedData[lindex] = absolute_difference;
        __syncthreads();

        // Parallel reduction
        for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
            if (lindex < offset) {
                sharedData[lindex] += sharedData[lindex + offset];
            }
            __syncthreads();
        }

        // Add the results in each block
        if (lindex == 0) {
            atomicAdd(&result[batch_id], sharedData[0]);
        }
    }
}